#include <iostream>
#include <vector>
#include <utility>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <hipsparse.h>

using namespace std;
namespace odeint = boost::numeric::odeint;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;

struct ham_lattice {
    value_type beta;
    hipsparseHandle_t   handle;
    hipsparseMatDescr_t descr;
    hipsparseHybMat_t   A;

    ham_lattice( value_type beta, hipsparseHandle_t handle,
	    hipsparseMatDescr_t descr, hipsparseHybMat_t A
	    ) : beta(beta) , descr(descr), handle(handle), A(A) { }

    struct scaled_pow3_functor {
        value_type beta;

        scaled_pow3_functor( value_type beta ) : beta(beta) {}

        __host__ __device__ value_type operator()( value_type q ) const {
	    return beta * q * q * q;
        }
    };

    void operator()( const state_type &q , state_type &dp ) const
    {
	static value_type one = 1;

	thrust::transform(q.begin(), q.end(), dp.begin(),
		scaled_pow3_functor(-beta));

	hipsparseDhybmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		&one, descr, A,
		thrust::raw_pointer_cast(&q[0]), &one,
		thrust::raw_pointer_cast(&dp[0])
		);
    }

};

struct index_modulus {
    int N;

    index_modulus(int n) : N(n) {}

    inline int operator()(int idx) const {
	if( idx <  0 ) return idx + N;
	if( idx >= N ) return idx - N;
	return idx;
    }
};


int main( int argc , char **argv )
{
    size_t n1 = argc > 1 ? atoi(argv[1]) : 64;
    size_t n2 = n1;

    size_t n = n1 * n2;
    value_type K = 0.1;
    value_type beta = 0.01;
    value_type t_max = 100.0;
    value_type dt = 0.01;

    std::vector<value_type> disorder( n );
    std::generate( disorder.begin(), disorder.end(), drand48 );

    // Create CUSPARSE matrix.
    hipsparseHandle_t   handle;
    hipsparseMatDescr_t descr;
    hipsparseHybMat_t   hyb;

    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseCreateHybMat(&hyb);

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    {
	std::vector< double > val;
	std::vector< int > col;
	std::vector< int > row;

	size_t N = n1 * n2;

	val.reserve(N * 5);
	col.reserve(N * 5);
	row.reserve(N + 1);

	index_modulus index(N);

	row.push_back( 0 );
	for( int i=0 ; i < n1 ; ++i ) {
	    for( int j=0 ; j < n2 ; ++j ) {
		row.push_back( row.back() + 5 );
		int idx = i * n2 + j;
		int is[5] = { idx , index( idx + 1 ) , index( idx - 1 ) , index( idx - n2 ) , index( idx + n2 ) };
		std::sort( is , is + 5 );
		for( int i=0 ; i < 5 ; ++i ) {
		    col.push_back( is[i] );
		    if( is[i] == idx ) val.push_back( - disorder[idx]  - 4.0 * K );
		    else val.push_back( K );
		}
	    }
	}

	thrust::device_vector<int>    dev_row(row);
	thrust::device_vector<int>    dev_col(col);
	thrust::device_vector<double> dev_val(val);

	hipsparseDcsr2hyb(handle, N, N, descr,
		thrust::raw_pointer_cast(&dev_val[0]),
		thrust::raw_pointer_cast(&dev_row[0]),
		thrust::raw_pointer_cast(&dev_col[0]),
		hyb, 5, HIPSPARSE_HYB_PARTITION_AUTO
		);
    }

    std::pair<state_type, state_type> X(
	    state_type( n1 * n2 ),
	    state_type( n1 * n2 )
	    );
    thrust::fill(X.first.begin(),  X.first.end(),  0);
    thrust::fill(X.second.begin(), X.second.end(), 0);
    X.first[ n1/2*n2+n2/2 ] = 1.0;


    odeint::symplectic_rkn_sb3a_mclachlan<
        state_type , state_type , value_type , state_type , state_type , value_type ,
        odeint::thrust_algebra , odeint::thrust_operations
        > stepper;

    odeint::integrate_const( stepper , ham_lattice(beta , handle, descr, hyb),
	    X, value_type(0.0), t_max, dt );


    std::vector< value_type > x1( n ) , p1( n );
    thrust::copy( X.first.begin(),  X.first.end(),  x1.begin() );
    thrust::copy( X.second.begin(), X.second.end(), p1.begin() );

    cout << x1[0] << "\t" << p1[0] << std::endl;
}
