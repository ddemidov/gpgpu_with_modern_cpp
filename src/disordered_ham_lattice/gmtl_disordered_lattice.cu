// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

// nvcc can't handle C++11 -> use good ole rand()

#include <iostream>
#include <cstdlib>
#include <string>
#include <cmath>
#include <utility>
#include <boost/numeric/mtl/mtl.hpp>

#include <boost/numeric/mtl/interface/odeint.hpp>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/algebra/vector_space_algebra.hpp>

#include <boost/timer.hpp>


namespace odeint = boost::numeric::odeint;

struct index_modulus 
{
    int N;

    index_modulus(int n) : N(n) {}

    inline int operator()(int idx) const {
	if( idx <  0 ) return idx + N;
	if( idx >= N ) return idx - N;
	return idx;
    }
};

template <typename value_type, typename Matrix>
struct disordered_lattice
{
    typedef mtl::dense_vector<value_type>         state_type;

    // v is kept outside the functor to avoid copy constructor calls
    disordered_lattice(value_type beta, const Matrix& A, state_type& v) 
      : beta(beta), A(A), v(v) { }

    void operator()(const state_type& q, state_type& dp) 
    {
	// compute product explicitly since implicit calculation causes expensive hipMalloc/-Free (yet)
	v= A * q;
	dp= -beta * q * q * q + v;
    }

  private:
    const value_type   beta;
    const Matrix&      A;
    state_type&        v;
};




int main(int argc, char* argv[])
{
    using namespace mtl;
    mtl::vampir_trace<9999>                            tracer;

    typedef double                                     value_type;
    typedef unsigned                                   size_type;
    typedef matrix::parameters<row_major, mtl::index::c_index, non_fixed::dimensions, false, size_type> para;
    typedef mtl::compressed2D<value_type, para>              matrix_type;
    typedef typename disordered_lattice<value_type, matrix_type>::state_type  state_type;

    size_type n1 = argc > 1 ? atoi(argv[1]) : 512 /* 64 */, n2= n1, n= n1 * n2;
    const value_type K = 0.1, beta = 0.01, dt= 0.01, t_max= 100.0;

    std::vector<value_type> disorder( n );
    std::generate( disorder.begin(), disorder.end(), drand48 );

    index_modulus index(n);
    matrix_type A(n, n);
    {
	mtl::matrix::inserter<matrix_type> ins(A);
	for( int i=0 ; i < n1 ; ++i ) 
	    for( int j=0 ; j < n2 ; ++j ) {
		int idx = i * n2 + j; 
		ins[idx][idx] << -disorder[idx] - 4.0 * K;
		ins[idx][index(idx + 1)] << K;
		ins[idx][index(idx - 1)] << K;
		ins[idx][index(idx + n2)] << K;
		ins[idx][index(idx - n2)] << K;
	    }
    }

    std::pair<state_type, state_type> X= std::make_pair(state_type(n, 0.0), state_type(n, 0.0));
    X.first[ n1/2 * n2 + n2/2 ]= 1.0;

    odeint::symplectic_rkn_sb3a_mclachlan<
        state_type, state_type, value_type, state_type, state_type, value_type,
        odeint::vector_space_algebra , odeint::default_operations
        > stepper;

    state_type   v(num_rows(A)); 
    disordered_lattice<value_type, matrix_type> sys( beta, A, v);
    boost::timer timer;
    odeint::integrate_const(stepper, sys, X, value_type(0.0), t_max, dt);
    hipDeviceSynchronize();
    std::cout << "Integration took " << timer.elapsed() << " s\n";
    mtl::irange rr(10);
 
    if (n1 == 512) {
	mtl::dense_vector<value_type> compare_first(10), compare_second(10);
	bool check= false;

	if (t_max == 10.0) {
	    compare_first= 0.695523,-1.06336,-0.301946,-0.923554,0.62739,3.06804,2.0448,0.0154956,2.0136,-0.775507;
	    compare_second= -1.17204,-2.89465,-0.679947,0.520701,1.06731,-2.01476,-1.60585,0.446732,-3.20896,-0.801548;
	    check= true;
	} else if (t_max > 0.0999 && t_max < 0.10001) {
	    compare_first= 2.98606,2.99274,2.98692,2.98669,2.98499,2.99569,2.99362,2.98714,2.99448,2.99035;
	    compare_second= -0.278452,-0.145232,-0.261421,-0.26603,-0.299831,-0.0862731,-0.127468,-0.256972,-0.110283,-0.192934;
	    check= true;
	}

	if (check) {
	    compare_first-= X.first[rr];
	    compare_second-= X.second[rr]; 

	    if (two_norm(compare_first) > 0.01 || two_norm(compare_second) > 0.01) {
		std::cerr << "Wrong result" << std::endl;
		// return 1;
	    }
	}
    }

    std::cout << X.first[0] << " " << X.second[0] << std::endl;
    return 0;
}

