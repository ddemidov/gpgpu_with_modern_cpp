#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#define DECORATE_CALLS
#include <boost/numeric/odeint.hpp>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

#include <stdio.h>
#include <hip/hip_runtime.h>

namespace odeint = boost::numeric::odeint;



//---------------------------------------------------------------------------
template <typename T>
struct point3d {
    T x;
    T y;
    T z;
};

template <typename T>
__host__ __device__ point3d<T> operator+(point3d<T> a, point3d<T> b) {
    point3d<T> c = {a.x + b.x, a.y + b.y, a.z + b.z};
    return c;
}

template< typename T>
__host__ __device__ point3d<T> operator+(T a, point3d<T> b)
{
    point3d<T> c={a + b.x, a + b.y, a + b.z};
    return c;
}

template< typename T>
__host__ __device__ point3d<T> operator+(point3d<T> b, T a)
{
    point3d<T> c={a + b.x, a + b.y, a + b.z};
    return c;
}


template <typename T>
__host__ __device__ point3d<T> operator-(point3d<T> a, point3d<T> b) {
    point3d<T> c = {a.x - b.x, a.y - b.y, a.z - b.z};
    return c;
}

template <typename T>
__host__ __device__ point3d<T> operator*(T a, point3d<T> b) {
    point3d<T> c = {a * b.x, a * b.y, a * b.z};
    return c;
}

template <typename T>
__host__ __device__ point3d<T> operator/(point3d<T> a, point3d<T> b) {
    point3d<T> c = {a.x / b.x, a.y / b.y, a.z / b.z};
    return c;
}


template <typename T>
__host__ __device__ point3d<T> abs(point3d<T> p)
{
    point3d<T> ret;
    ret.x = abs( p.x );
    ret.y = abs( p.y );
    ret.z = abs( p.z );
    return ret;
}

namespace boost { namespace numeric { namespace odeint { 
template<typename T>
struct is_resizeable< point3d<T> > : boost::false_type { };
} } } 

namespace boost { namespace numeric { namespace odeint { 
template<typename T>
struct vector_space_reduce< point3d< T > >
{
    template< class Op >
    __host__ __device__ T operator()( const point3d<T> &x , Op op , T init ) const
    {
        init = op( init , x.x );
        init = op( init , x.y );
        init = op( init , x.z );
        return init;
    }
};
} } } 
//---------------------------------------------------------------------------



typedef double value_type;
typedef point3d<value_type> state_type;

//---------------------------------------------------------------------------
std::ostream& operator<<(std::ostream &os, const state_type &s) {
    return os << "[" << s.x << " " << s.y << " " << s.z << "]";
}


const value_type sigma = 10.0;
const value_type b = 8.0 / 3.0;
const value_type t_max = 1.0;

struct lorenz_system {
    value_type R;
    
    lorenz_system(value_type r = 0) : R(r) {}

    __host__ __device__ void operator()(const state_type &s, state_type &dsdt, value_type t) {
	dsdt.x = sigma * (s.y - s.x);
	dsdt.y = R * s.x - s.y - s.x * s.z;
	dsdt.z = s.x * s.y - b * s.z;
    }
};

struct stepper_functor
{

    odeint::controlled_runge_kutta<
        odeint::runge_kutta_cash_karp54_classic<
	    state_type, value_type, state_type, value_type,
	    odeint::vector_space_algebra, odeint::default_operations,
	    odeint::never_resizer
	    > > stepper;

//    value_type t , dt;

    stepper_functor( void ) /*: t( 0.0 ) , dt( 0.01 ) */ { }

    template <class T>
    __host__ __device__ void operator()(T s)
    {
        using namespace odeint;

        state_type    &state = thrust::get<0>(s);
        lorenz_system &sys   = thrust::get<1>(s);
        value_type &t = thrust::get<2>(s);
        value_type &dt = thrust::get<3>(s);
        

        const size_t max_attempts = 1000;

        size_t count = 0;
        while( t < t_max )
        {
            if( t_max < ( t + dt ) )
            {
                dt = t_max - t;
            }

            size_t trials = 0;
            controlled_step_result res = success;
            do
            {
                res = stepper.try_step( sys , state , t , dt );
                ++trials;
            }
            while( ( res == fail ) && ( trials < max_attempts ) );
            if( trials == max_attempts ) break;
            ++count;
        }
    }
};



//---------------------------------------------------------------------------
int main(int argc, char *argv[])
{
    using namespace std;

    size_t n = argc > 1 ? atoi(argv[1]) : 1024;

    std::vector<lorenz_system> ensemble_host(n);
    value_type Rmin = 0.1 , Rmax = 50.0 , dR = ( n > 1 ) ? ( Rmax - Rmin ) / value_type( n - 1 ) : 0.0;
    for( size_t i=0 ; i<n ; ++i )
        ensemble_host[i] = lorenz_system(Rmin + dR * value_type( i ));
    thrust::device_vector<lorenz_system> ensemble = ensemble_host;

    state_type seed = {10, 10, 10};
    thrust::device_vector<state_type> x(n);
    thrust::device_vector< value_type > t(n) , dt(n);
    thrust::fill(x.begin(), x.end(), seed);
    thrust::fill(t.begin(), t.end(), 0.0);
    thrust::fill(dt.begin(), dt.end(), 0.01);


    stepper_functor step;
    thrust::for_each(
        thrust::make_zip_iterator(
            thrust::make_tuple(x.begin(), ensemble.begin(), t.begin(), dt.begin() )),
        thrust::make_zip_iterator(
            thrust::make_tuple(x.end(), ensemble.end(), t.end(), dt.end() )),
        step);

//        odeint::integrate_const(stepper, std::ref(sys[i]), X[i], double(0), t_max, dt);



    for( size_t i=0 ; i<n ; ++i )
        std::cout << ensemble_host[i].R << "\t" << x[i] << "\t" << t[i] << "\t" << dt[i] << std::endl;




    // DEBUG STUFF
    //
    // lorenz_system l( 28.0 );
    // odeint::controlled_runge_kutta<
    //     odeint::runge_kutta_cash_karp54_classic<
    //         state_type, value_type, state_type, value_type,
    //         odeint::vector_space_algebra, odeint::default_operations,
    //         odeint::never_resizer
    //         > > stepper2;
    // state_type xx = { 10.0 , 10.0 , 10.0 };
    // double t = 0.0 , dt = 0.01;
    // odeint::controlled_step_result res = stepper2.try_step( l , xx , t , dt );
    // cout << 28.0 << "\t" << t << "\t" << dt << "\t" << int( res) << "\t" << xx << endl;


    // state_type x_old = { 10.0 , 10.0 , 10.0 } , x_new;
    // state_type dxdt_old;
    // state_type x_err;
    // value_type dt = 0.01;
    // lorenz_system l( 28.0 );


    // odeint::runge_kutta_cash_karp54_classic<
    //     state_type, value_type, state_type, value_type,
    //     odeint::vector_space_algebra, odeint::default_operations,
    //     odeint::never_resizer
    //     > stepper;

    // l( x_old , dxdt_old , 0.0 );
    // stepper.do_step( l , x_old , dxdt_old , 0.0 , x_new , dt , x_err );

    // // cout << x_old << endl;
    // // cout << dxdt_old << endl;
    // // cout << x_new << endl;
    // // cout << x_err << endl;
    
    // // value_type eps_abs = 1.0e-6 , eps_rel = 1.0e-6 , a_x = 1.0 , a_dxdt = 1.0;

    // // using namespace odeint;
    // // vector_space_algebra algebra;
    // // algebra.for_each3( x_err , x_old , dxdt_old ,
    // //                    default_operations::rel_error< value_type >( eps_abs , eps_rel , a_x , a_dxdt * dt ) );

    // // value_type res = algebra.reduce( x_err , default_operations::maximum< value_type >() , 0.0 );

    // // cout << x_err << endl;
    // // cout << res << endl;

    // odeint::controlled_runge_kutta<
    //     odeint::runge_kutta_cash_karp54_classic<
    //         state_type, value_type, state_type, value_type,
    //         odeint::vector_space_algebra, odeint::default_operations,
    //         odeint::never_resizer
    //         > > stepper2;


    // double t = 0.0;
    // odeint::controlled_step_result res = stepper2.try_step( l , x_old , dxdt_old , t , x_new , dt );

    // cout << int( res ) << " " << t << " " << dt << endl;
    // cout << x_old << endl;
    // cout << dxdt_old << endl;
    // cout << x_new << endl;

}
