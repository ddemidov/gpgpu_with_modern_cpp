#include <iostream>
#include <cmath>
#include <utility>
#include <cstdlib>


#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>

#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>



using namespace std;
using namespace boost::numeric::odeint;


typedef double value_type;

typedef thrust::device_vector< value_type > state_type;

const value_type sigma = 10.0;
const value_type b = 8.0 / 3.0;


struct lorenz_system
{
    struct lorenz_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t ) const
        {
            value_type R = thrust::get< 3 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
            value_type z = thrust::get< 2 >( t );
            thrust::get< 4 >( t ) = sigma * ( y - x );
            thrust::get< 5 >( t ) = R * x - y - x * z;
            thrust::get< 6 >( t ) = -b * z + x * y ;

        }
    };

    lorenz_system( size_t N , const state_type &beta )
    : m_N( N ) , m_beta( beta ) { }

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) ,
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
                        m_beta.begin() ,
                        boost::begin( dxdt ) ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N  ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
                        boost::begin( x ) + 3 * m_N ,
                        m_beta.begin() ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N ,
                        boost::begin( dxdt ) + 3 * m_N  ) ) ,
                lorenz_functor() );
    }

    size_t m_N;
    const state_type &m_beta;
};


size_t N;
const value_type dt = 0.01;
const value_type t_max = 100.0;


int main( int argc , char* argv[] )
{
    // int driver_version , runtime_version;
    // hipDriverGetVersion( &driver_version );
    // hipRuntimeGetVersion ( &runtime_version );
    // cout << driver_version << "\t" << runtime_version << endl;

    N = argc > 1 ? atoi(argv[1]) : 1024;

    vector< value_type > beta_host( N );
    const value_type beta_min = value_type(0.1) , beta_max = value_type(50.0);
    for( size_t i=0 ; i<N ; ++i )
        beta_host[i] = beta_min + value_type( i ) * ( beta_max - beta_min ) / value_type( N - 1 );

    state_type beta = beta_host;


    //[ thrust_lorenz_parameters_integration
    state_type x( 3 * N );

    // initialize x,y,z
    thrust::fill( x.begin() , x.end() , value_type(10.0) );



    typedef runge_kutta4< state_type , value_type , state_type , value_type ,
			  thrust_algebra , thrust_operations > stepper_type;


    lorenz_system lorenz( N , beta );
    integrate_const( stepper_type() , lorenz , x , value_type(0.0) , t_max , dt );

    thrust::host_vector< value_type > res = x;
    // for( size_t i=0 ; i<N ; ++i ) cout << res[i] << "\t" << beta_host[i] << "\n";
    cout << res[0] << endl;



    return 0;
}
