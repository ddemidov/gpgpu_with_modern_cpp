#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <utility>
#include <cstdlib>
#include <algorithm>


#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>

#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>



using namespace std;
using namespace boost::numeric::odeint;


typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
// typedef thrust::host_vector< value_type > state_type;
// typedef thrust::host_vector< size_t > index_vector_type;


struct oscillator
{
    value_type m_omega;
    value_type m_amp;
    value_type m_offset;
    value_type m_omega_d;
    size_t m_N;

    struct oscillator_functor
    {
        value_type m_eps;
        value_type m_omega;
        oscillator_functor( value_type omega , value_type eps )
            : m_omega( omega ) , m_eps( eps ) { }

        template< class T >
        __host__ __device__
        void operator()( T t ) const
        {
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
            thrust::get< 2 >( t ) =  m_omega * y + m_eps * x;
            thrust::get< 3 >( t ) = -m_omega * x + m_eps * y;
        }
    };


    oscillator( size_t N , double omega = 1.0 , double amp = 0.5 , double offset = 0.0 , double omega_d = 1.2 )
        : m_N( N ) , m_omega( omega ) , m_amp( amp ) , m_offset( offset ) , m_omega_d( omega_d ) { }

    void operator()( const state_type &x , state_type &dxdt , double t ) const
    {
        double eps = m_offset + m_amp * cos( m_omega_d * t );
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    boost::begin( x ) ,
                    boost::begin( x ) + m_N ,
                    boost::begin( dxdt ) ,
                    boost::begin( dxdt ) + m_N 
                    ) ) ,
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    boost::begin( x ) + m_N ,
                    boost::begin( x ) + 2 * m_N ,
                    boost::begin( dxdt ) + m_N ,
                    boost::begin( dxdt ) + 2 * m_N
                    ) ) ,
            oscillator_functor( m_omega , eps ) );
    }
};



size_t N;
const value_type dt = 0.01;
const value_type t_max = 100.0;


int main( int argc , char* argv[] )
{
    // int driver_version , runtime_version;
    // hipDriverGetVersion( &driver_version );
    // hipRuntimeGetVersion ( &runtime_version );
    // cout << driver_version << "\t" << runtime_version << endl;

    N = argc > 1 ? atoi(argv[1]) : 1024;


    std::vector<value_type> x( 2 * N );
    std::generate( x.begin() , x.end() , drand48 );

    state_type X( 2 * N );
    thrust::copy( x.begin() , x.end() , X.begin() );


    typedef runge_kutta4< state_type , value_type , state_type , value_type ,
			  thrust_algebra , thrust_operations > stepper_type;
    integrate_const( stepper_type() , oscillator( 1.0 , 0.2 , 0.0 , 1.2 ) , X , value_type(0.0) , t_max , dt );

    thrust::host_vector< value_type > res = X;
    // for( size_t i=0 ; i<N ; ++i ) cout << res[i] << "\t" << beta_host[i] << "\n";
    cout << res[0] << endl;


    return 0;
}
