#include "hip/hip_runtime.h"
// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

// nvcc can't handle C++11 -> use good ole rand()

#include <cassert>
#include <iostream>
#include <cstdlib>
#include <string>
#include <cmath>
#include <utility>
#include <boost/numeric/mtl/mtl.hpp>

#include <boost/numeric/mtl/interface/odeint.hpp>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/algebra/vector_space_algebra.hpp>

#include <boost/timer.hpp>


namespace odeint = boost::numeric::odeint;

using namespace mtl;

struct stencil_kernel
{
    static const int start= -1, end= 1;

    template <typename Vector>
    __device__ double inner_stencil(const Vector& v, int i) const
    {
	return v[i-1] + v[i] + v[i+1];
    }

    template <typename Vector>
    __device__ double outer_stencil(const Vector& v, int i, int n) const
    {
	double s= v[i];
	if (i > 0) s+= v[i-1];
	if (i+1 < n) s+= v[i+1];
	return s;
    }
};

template <typename Stencil>
struct test_kernel
{
    test_kernel(const dense_vector<double>& v, dense_vector<double>& w, Stencil stencil) 
      : v(v), wp(w.device_data), n(mtl::vector::size(v)), stencil(stencil) {}

    __device__ void operator()()
    {
	const int size= (0x4000 - 0x28) / sizeof(double);

	__shared__ double tmp[size];
	const unsigned tid= threadIdx.x, bs= blockDim.x;

	for (int i= tid; i < size; i+= bs)
	    tmp[i]= v.dat(i);
	__syncthreads();

	// for (int i= tid; i < n; i+= bs)
	//     stencil.inner_stencil(tmp, i);


#if 1
	wp[tid]= stencil.outer_stencil(tmp, tid, n);

	for (int i= tid + bs; i < n - bs; i+= bs)
	    wp[i]= stencil.inner_stencil(tmp, i);

	wp[n - bs + tid]= stencil.outer_stencil(tmp, n - bs + tid, n);
#endif
    }

    vector::device_expr<dense_vector<double> > v;
    double*                                    wp;
    int                                        n;
    Stencil                                    stencil;
};



int main(int argc, char* argv[])
{
    using namespace mtl;
    mtl::vampir_trace<9999>                            tracer;

    dense_vector<double> v(128), w(128);
    iota(v);

    v.to_device();
    w.to_device();
    test_kernel<stencil_kernel> k(v, w, stencil_kernel());
    launch_function<<<1, 32>>>(k);
    w.to_host();

    std::cout << "w is " << w /*[irange(10)]*/ << '\n';

    return 0;
}

