#include "hip/hip_runtime.h"
/*
 * phase_osc_chain.cu
 *
 *  Created on: Apr 1, 2011
 *      Author: mario
 */

/*
 * This example shows how to use odeint on CUDA devices with thrust.
 * Note that we require at least Version 3.2 of the nVidia CUDA SDK
 * and the thrust library should be installed in the CUDA include
 * folder.
 *
 * As example we use a chain of phase oscillators with nearest neighbour
 * coupling, as described in:
 *
 * Avis H. Cohen, Philip J. Holmes and Richard H. Rand:
 * JOURNAL OF MATHEMATICAL BIOLOGY Volume 13, Number 3, 345-369,
 *
 */

#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace std;

using namespace boost::numeric::odeint;


//change this to float if your device does not support double computation
typedef double value_type;


typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;


class phase_oscillators
{

public:

    struct sys_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )  // this functor works on tuples of values
        {
            // first, unpack the tuple into value, neighbors and omega
            const value_type phi = thrust::get<0>(t);
            const value_type phi_left = thrust::get<1>(t);  // left neighbor
            const value_type phi_right = thrust::get<2>(t); // right neighbor
            const value_type omega = thrust::get<3>(t);
            // the dynamical equation
            thrust::get<4>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );
        }
    };

    phase_oscillators( const state_type &omega )
        : m_omega( omega ) , m_N( omega.size() ) , m_prev( omega.size() ) , m_next( omega.size() )
    {
        // build indices pointing to left and right neighbours
        thrust::counting_iterator<size_t> c( 0 );
        thrust::copy( c , c+m_N-1 , m_prev.begin()+1 );
        m_prev[0] = 0; // m_prev = { 0 , 0 , 1 , 2 , 3 , ... , N-1 }

        thrust::copy( c+1 , c+m_N , m_next.begin() );
        m_next[m_N-1] = m_N-1; // m_next = { 1 , 2 , 3 , ... , N-1 , N-1 }
    }

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.begin() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.begin() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.begin() ) ,
                                m_omega.begin() ,
                                dxdt.begin()
                                ) ),
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.end() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.end() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.end() ) ,
                                m_omega.end() ,
                                dxdt.end()) ) ,
                sys_functor()
                );
    }

private:

    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;
};





size_t n;
const value_type dt = 0.01;
const value_type t_max = 100.0;

int main( int argc , char* argv[] )
{
    n = ( argc > 1 ) ? atoi(argv[1]) : 1024;
    const value_type epsilon = 6.0 / ( n * n ); // should be < 8/N^2 to see phase locking

    vector< value_type > x_host( n );
    vector< value_type > omega_host( n );
    for( size_t i=0 ; i<n ; ++i )
    {
        x_host[i] = 2.0 * M_PI * drand48();
        omega_host[i] = double( n - i ) * epsilon; // decreasing frequencies
    }

    state_type x = x_host;
    state_type omega = omega_host;

    runge_kutta4< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper;

    phase_oscillators sys( omega );

    integrate_const( stepper , sys , x , 0.0 , t_max , dt );

    std::vector< value_type > res( n );
    thrust::copy( x.begin() , x.end() , res.begin() );
    cout << res[0] << endl;
}
