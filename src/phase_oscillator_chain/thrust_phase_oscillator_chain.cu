#include "hip/hip_runtime.h"
/*
 * phase_osc_chain.cu
 *
 *  Created on: Apr 1, 2011
 *      Author: mario
 */

/*
 * This example shows how to use odeint on CUDA devices with thrust.
 * Note that we require at least Version 3.2 of the nVidia CUDA SDK
 * and the thrust library should be installed in the CUDA include
 * folder.
 *
 * As example we use a chain of phase oscillators with nearest neighbour
 * coupling, as described in:
 *
 * Avis H. Cohen, Philip J. Holmes and Richard H. Rand:
 * JOURNAL OF MATHEMATICAL BIOLOGY Volume 13, Number 3, 345-369,
 *
 */

#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace std;
using namespace boost::numeric::odeint;


//change this to float if your device does not support double computation
typedef double value_type;


typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;


class phase_oscillators
{

public:

    struct sys_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )  // this functor works on tuples of values
        {
            // first, unpack the tuple into value, neighbors and omega
            const value_type phi = thrust::get<0>(t);
            const value_type phi_left = thrust::get<1>(t);  // left neighbor
            const value_type phi_right = thrust::get<2>(t); // right neighbor
            const value_type omega = thrust::get<3>(t);
            // the dynamical equation
            thrust::get<4>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );
        }
    };

    phase_oscillators( const state_type &omega )
        : m_omega( omega ) , m_N( omega.size() )
    {
    }

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.begin() + 1,
                                x.begin(),
                                x.begin() + 2,
                                m_omega.begin() ,
                                dxdt.begin()
                                ) ),
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.end() - 1,
                                x.end() - 2,
                                x.end(),
                                m_omega.end() ,
                                dxdt.end()) ) ,
                sys_functor()
                );

        dxdt[0] = dxdt[1];
        dxdt[m_N + 1] = dxdt[m_N];
    }

private:

    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;
};





size_t n;
const value_type dt = 0.01;
const value_type t_max = 100.0;

int main( int argc , char* argv[] )
{
    n = ( argc > 1 ) ? atoi(argv[1]) : 1024;
    const value_type epsilon = 6.0 / ( n * n ); // should be < 8/N^2 to see phase locking

    vector< value_type > x_host( n + 2 );
    vector< value_type > omega_host( n );
    for( size_t i=0 ; i<n ; ++i )
    {
        x_host[i + 1] = 2.0 * M_PI * drand48();
        omega_host[i] = double( n - i ) * epsilon; // decreasing frequencies
    }

    x_host[0] = x_host[1];
    x_host[n + 1] = x_host[n];

    state_type x = x_host;
    state_type omega = omega_host;

    runge_kutta4< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper;

    phase_oscillators sys( omega );

    integrate_const( stepper , sys , x , 0.0 , t_max , dt );

    std::vector< value_type > res( n );
    thrust::copy( x.begin() + 1, x.end() - 1, res.begin() );
    cout << res[0] << endl;
}
