#include "hip/hip_runtime.h"
// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

// nvcc can't handle C++11 -> use good ole rand()

#include <cassert>
#include <iostream>
#include <cstdlib>
#include <string>
#include <cmath>
#include <utility>
#include <boost/numeric/mtl/cuda/cuda_utility.hpp>

#include <boost/numeric/mtl/mtl.hpp>

#include <boost/numeric/mtl/interface/odeint.hpp>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/algebra/vector_space_algebra.hpp>

#include <boost/timer.hpp>


namespace odeint = boost::numeric::odeint;

using namespace mtl;


struct stencil_kernel
{
    typedef double value_type;
    static const int start= -1, end= 1;

    stencil_kernel(int n) : n(n) {}

    template <typename Vector>
    __device__ __host__ value_type operator()(const Vector& v, int i) const
    {
	return sin(v[i+1] - v[i]) + sin(v[i] - v[i-1]);
    }

    template <typename Vector>
    __device__ __host__ value_type outer_stencil(const Vector& v, int i, int offset= 0) const
    {
	value_type s1= i > offset? sin(v[i] - v[i-1]) : sin(v[i]), 
	           s2= i+1 < n + offset? sin(v[i+1] - v[i]) : sin(v[i]);
	return s1 + s2;
    }

    int n;
};

template <typename State>
struct sys_func
{
    typedef typename Collection<State>::value_type value_type;

    sys_func(const State& omega) 
      : omega(omega), S(num_rows(omega)) {}

    void operator()(const State &x, State &dxdt, value_type t) const
    {
	dxdt = S * x;
        dxdt += omega;
    }

    const State&   omega;
    mtl::matrix::stencil1D<stencil_kernel> S; 
};


int main(int argc, char* argv[])
{
    using namespace mtl;
    mtl::vampir_trace<9999>                            tracer;

    typedef double                    value_type;
    typedef dense_vector<value_type>  state_type;

    const value_type dt= 0.01, pi= M_PI, t_max= 100.0;
    const size_t n= argc > 1 ? atoi(argv[1]) : 1024;
    const value_type epsilon = 6.0 / ( n * n ); // should be < 8/N^2 to see phase locking

    state_type omega(n), x(n), tmp(n);
    for (size_t i= 0; i < n; ++i) {
        x[i] = 2.0 * pi * drand48();
        omega[i] = double(n - i) * epsilon; // decreasing frequencies
    }

    odeint::runge_kutta4<
	    state_type, value_type, state_type, value_type,
	    odeint::vector_space_algebra, odeint::default_operations
	    > stepper;

    sys_func<state_type> sys(omega);
    boost::timer timer;
    odeint::integrate_const(stepper, boost::ref(sys), x, 0.0, t_max, dt);
    hipDeviceSynchronize();
    std::cout << "Integration took " << timer.elapsed() << " s\n";
    
    std::cout << "Result is " << x[0] << '\n';

    return 0;
}

